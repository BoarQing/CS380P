#include "hip/hip_runtime.h"
#include "util.hpp"
#include <cfloat>
#include <hip/hip_runtime.h>

#define DIFF_TIME 4
__global__ void ComputeCentroidDiff(int input_len, int dim, int centroid_count,
                                    double *input, int *label, double *centroid,
                                    double *distance) {
  int input_index = blockIdx.x * DIFF_TIME + threadIdx.x / centroid_count;
  int centroid_index = threadIdx.x % centroid_count;

  if (input_index < input_len) {
    double *my_input = input + input_index * dim;
    double *my_centroid = centroid + centroid_index * dim;
    double total_diff = 0.0;
    for (int i = 0; i < dim; ++i) {
      double diff = my_input[i] - my_centroid[i];
      total_diff += diff * diff;
    }
    *(distance + input_index * centroid_count + centroid_index) = total_diff;
  }
}

#define THREAD_COUNT 64
__global__ void FindNearestCentroid(int input_len, int centroid_count,
                                    double *distance, int *label) {
  int idx = blockIdx.x * THREAD_COUNT + threadIdx.x;
  if (idx < input_len) {
    double *my_distance = distance + idx * centroid_count;
    double min_dis = my_distance[0];
    double min_idx = 0;
    for (int i = 1; i < centroid_count; ++i) {
      if (my_distance[i] < min_dis) {
        min_dis = my_distance[i];
        min_idx = i;
      }
    }
    label[idx] = min_idx;
  }
}

#define REDUCE_COUNT 32
__global__ void ReducedPartialAvgLabel(int input_len, int dim,
                                       int centroid_count, double *input,
                                       int *label, double *centroid,
                                       int *count) {
  int each_work_load = input_len / REDUCE_COUNT;
  int beg = blockIdx.x * each_work_load;
  int end = beg + each_work_load;
  if (input_len - end < each_work_load) {
    end = input_len;
  }
  int *my_count = count + blockIdx.x * centroid_count;
  double *my_centroid =
      centroid + blockIdx.x * dim * centroid_count + threadIdx.x;
  double *my_input = input + beg * dim + threadIdx.x;

  for (int i = beg; i < end; ++i) {
    int my_label = label[i];
    double *currrent_centroid = my_centroid + my_label * dim;
    *currrent_centroid += *my_input;
    my_input += dim;
    if (threadIdx.x == 0) {
      ++my_count[my_label];
    }
  }
}

__global__ void ReducedAvgLabel(int dim, int centroid_count, int *reduce_count,
                                double *reduce_centroid, double *centroid) {
  double sum = 0.0;
  int count = 0;
  double *out_centroid = centroid + blockIdx.x * dim + threadIdx.x;
  double *my_centroid = reduce_centroid + blockIdx.x * dim + threadIdx.x;
  int *my_count = reduce_count + blockIdx.x;

  for (int i = 0; i < REDUCE_COUNT; i++) {
    sum += *my_centroid;
    count += *my_count;
    my_centroid += centroid_count * dim;
    my_count += centroid_count;
  }
  *out_centroid = sum / count;
}

__global__ void Converge(int dim, double threshold, bool *converge, double *c1,
                         double *c2) {
  double *my_c1 = c1 + blockIdx.x * dim;
  double *my_c2 = c2 + blockIdx.x * dim;
  double total_diff = 0.0;
  for (int i = 0; i < dim; i++) {
    double diff = my_c1[i] - my_c2[i];
    total_diff += diff * diff;
  }
  converge[blockIdx.x] = total_diff < threshold;
}

bool IsConverged(int centroid_count, bool *converge) {
  for (int i = 0; i < centroid_count; i++) {
    if (converge[i] == false) {
      return false;
    }
  }
  return true;
}

KMeans_Ret KMeans(Arg *arg) {
  double *centroid = GetRandomCentroid(arg);
  int block_count = int(ceil((double)arg->input_len / THREAD_COUNT));
  int diff_count = int(ceil((double)arg->input_len / DIFF_TIME));
  int iteration = 0;
  bool done = false;
  double *device_centroid;
  size_t input_size = arg->dims * arg->input_len * sizeof(double);
  double *device_input;
  hipMalloc(&device_input, input_size);
  hipMemcpyAsync(device_input, arg->input, input_size, hipMemcpyHostToDevice);
  size_t centroid_size = sizeof(double) * arg->dims * arg->num_cluster;
  hipMalloc(&device_centroid, centroid_size * 2);
  hipMemcpyAsync(device_centroid, centroid, centroid_size,
                  hipMemcpyHostToDevice);
  size_t label_size = sizeof(int) * arg->input_len;
  int *label = (int *)malloc(label_size);
  int *device_label;
  hipMalloc(&device_label, label_size);
  size_t distance_size = sizeof(double) * arg->input_len * arg->num_cluster;
  double *device_distance;
  hipMalloc(&device_distance, distance_size);
  size_t converge_size = sizeof(bool) * arg->num_cluster;
  bool *device_converge;
  bool *converge = (bool *)malloc(converge_size);
  hipMalloc(&device_converge, converge_size);
  int *reduce_count;
  size_t reduce_count_size = sizeof(int) * arg->num_cluster * REDUCE_COUNT;
  hipMalloc(&reduce_count, reduce_count_size);
  double *reduce_centroid;
  size_t reduce_centroid_size =
      sizeof(double) * arg->num_cluster * arg->dims * REDUCE_COUNT;
  hipMalloc(&reduce_centroid, reduce_centroid_size);
  hipDeviceSynchronize(); // for async
  double *new_centroid = nullptr;
  TICK();
  while (!done) {
    double *old_centroid =
        device_centroid + arg->dims * arg->num_cluster * (iteration % 2);
    new_centroid =
        device_centroid + arg->dims * arg->num_cluster * ((iteration + 1) % 2);

    ComputeCentroidDiff<<<diff_count, arg->num_cluster * DIFF_TIME>>>(
        arg->input_len, arg->dims, arg->num_cluster, device_input, device_label,
        old_centroid, device_distance);
    FindNearestCentroid<<<block_count, THREAD_COUNT>>>(
        arg->input_len, arg->num_cluster, device_distance, device_label);
    hipMemset(reduce_count, 0, reduce_count_size);
    hipMemset(reduce_centroid, 0, reduce_centroid_size);
    ReducedPartialAvgLabel<<<REDUCE_COUNT, arg->dims>>>(
        arg->input_len, arg->dims, arg->num_cluster, device_input, device_label,
        reduce_centroid, reduce_count);
    ReducedAvgLabel<<<arg->num_cluster, arg->dims>>>(
        arg->dims, arg->num_cluster, reduce_count, reduce_centroid,
        new_centroid);
    Converge<<<arg->num_cluster, 1>>>(
        arg->dims, arg->threshold, device_converge, old_centroid, new_centroid);
    hipMemcpy(converge, device_converge, converge_size,
               hipMemcpyDeviceToHost);
    bool converged = IsConverged(arg->num_cluster, converge);
    ++iteration;
    done = iteration >= arg->max_num_iter || converged;
  }
  TOCK(iteration);
  hipMemcpyAsync(label, device_label, label_size, hipMemcpyDeviceToHost);
  hipMemcpyAsync(centroid, new_centroid, centroid_size,
                  hipMemcpyDeviceToHost);
  hipDeviceSynchronize(); // must finish before free!
  hipFree(device_distance);
  hipFree(device_input);
  hipFree(device_centroid);
  hipFree(device_label);
  hipFree(device_converge);
  hipFree(reduce_centroid);
  hipFree(reduce_count);
  return KMeans_Ret{centroid, label};
}

int main(int argc, char *argv[]) {
  Arg arg = GetArg(argc, argv);
  KMeans_Ret ret = KMeans(&arg);
  if (arg.control_flag) {
    PrintCentroid(&arg, ret.centroid);
  } else {
    PrintLabel(&arg, ret.label);
  }
  free(ret.centroid);
  free(ret.label);
  FreeArg(&arg);
  return 0;
}