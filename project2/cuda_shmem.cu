#include "hip/hip_runtime.h"
#include "util.hpp"
#include <cfloat>
#include <hip/hip_runtime.h>

typedef struct {
  double diff;
  int idx;
} Diff_Info;

#define REDUCE_COUNT 32
__global__ void ReducedFindNearestCentroid(int input_len, int dim,
                                           int centroid_count, double *input,
                                           double *centroid, int *label) {
  extern __shared__ Diff_Info share_diff[];
  double *my_centroid = centroid + dim * threadIdx.x;
  double *my_input = input + blockIdx.x * dim;
  double total_diff = 0.0;
  if (threadIdx.x < centroid_count) {
    for (int i = 0; i < dim; ++i) {
      double diff = my_input[i] - my_centroid[i];
      total_diff += diff * diff;
    }
  } else {
    total_diff = FLT_MAX;
  }
  share_diff[threadIdx.x].diff = total_diff;
  share_diff[threadIdx.x].idx = threadIdx.x;
  __syncthreads();
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    int next = threadIdx.x + s;
    if (threadIdx.x < s) {
      if (share_diff[threadIdx.x].diff > share_diff[next].diff) {
        share_diff[threadIdx.x].diff = share_diff[next].diff;
        share_diff[threadIdx.x].idx = share_diff[next].idx;
      }
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    label[blockIdx.x] = share_diff[0].idx;
  }
}

__global__ void ReducedPartialAvgLabel(int input_len, int dim,
                                       int centroid_count, double *input,
                                       int *label, double *centroid,
                                       int *count) {
  int each_work_load = input_len / REDUCE_COUNT;
  int beg = blockIdx.x * each_work_load;
  int end = beg + each_work_load;
  if (input_len - end < each_work_load) {
    end = input_len;
  }
  int *my_count = count + blockIdx.x * centroid_count;
  double *my_centroid =
      centroid + blockIdx.x * dim * centroid_count + threadIdx.x;
  double *my_input = input + beg * dim + threadIdx.x;

  for (int i = beg; i < end; ++i) {
    int my_label = label[i];
    double *currrent_centroid = my_centroid + my_label * dim;
    *currrent_centroid += *my_input;
    my_input += dim;
    if (threadIdx.x == 0) {
      ++my_count[my_label];
    }
  }
}

__global__ void ReducedAvgLabel(int dim, int centroid_count, int *reduce_count,
                                double *reduce_centroid, double *centroid) {
  double sum = 0.0;
  int count = 0;
  double *out_centroid = centroid + blockIdx.x * dim + threadIdx.x;
  double *my_centroid = reduce_centroid + blockIdx.x * dim + threadIdx.x;
  int *my_count = reduce_count + blockIdx.x;

  for (int i = 0; i < REDUCE_COUNT; i++) {
    sum += *my_centroid;
    count += *my_count;
    my_centroid += centroid_count * dim;
    my_count += centroid_count;
  }
  *out_centroid = sum / count;
}

__global__ void Converge(int dim, double threshold, bool *converge, double *c1,
                         double *c2) {
  double *my_c1 = c1 + blockIdx.x * dim;
  double *my_c2 = c2 + blockIdx.x * dim;
  double total_diff = 0.0;
  for (int i = 0; i < dim; i++) {
    double diff = my_c1[i] - my_c2[i];
    total_diff += diff * diff;
  }
  converge[blockIdx.x] = total_diff < threshold;
}

bool IsConverged(int centroid_count, bool *converge) {
  for (int i = 0; i < centroid_count; i++) {
    if (converge[i] == false) {
      return false;
    }
  }
  return true;
}

int GetRoundUpValue(int x) {
  int base = 1;
  while (base < x) {
    base *= 2;
  }
  return base;
}

KMeans_Ret KMeans(Arg *arg) {
  double *centroid = GetRandomCentroid(arg);
  int iteration = 0;
  bool done = false;
  double *device_centroid;
  size_t input_size = arg->dims * arg->input_len * sizeof(double);
  double *device_input;
  hipMalloc(&device_input, input_size);
  hipMemcpyAsync(device_input, arg->input, input_size, hipMemcpyHostToDevice);
  size_t centroid_size = sizeof(double) * arg->dims * arg->num_cluster;
  hipMalloc(&device_centroid, centroid_size * 2);
  hipMemcpyAsync(device_centroid, centroid, centroid_size,
                  hipMemcpyHostToDevice);
  size_t label_size = sizeof(int) * arg->input_len;
  int *label = (int *)malloc(label_size);
  int *device_label;
  hipMalloc(&device_label, label_size);
  size_t converge_size = sizeof(bool) * arg->num_cluster;
  bool *device_converge;
  bool *converge = (bool *)malloc(converge_size);
  hipMalloc(&device_converge, converge_size);
  int *reduce_count;
  size_t reduce_count_size = sizeof(int) * arg->num_cluster * REDUCE_COUNT;
  hipMalloc(&reduce_count, reduce_count_size);
  double *reduce_centroid;
  size_t reduce_centroid_size =
      sizeof(double) * arg->num_cluster * arg->dims * REDUCE_COUNT;
  hipMalloc(&reduce_centroid, reduce_centroid_size);
  hipDeviceSynchronize(); // for async
  int round_up_centroid_count = GetRoundUpValue(arg->num_cluster);
  double *new_centroid = nullptr;
  TICK();
  while (!done) {
    double *old_centroid =
        device_centroid + arg->dims * arg->num_cluster * (iteration % 2);
    new_centroid =
        device_centroid + arg->dims * arg->num_cluster * ((iteration + 1) % 2);
    ReducedFindNearestCentroid<<<arg->input_len, round_up_centroid_count,
                                 round_up_centroid_count * sizeof(Diff_Info)>>>(
        arg->input_len, arg->dims, arg->num_cluster, device_input, old_centroid,
        device_label);
    hipMemset(reduce_count, 0, reduce_count_size);
    hipMemset(reduce_centroid, 0, reduce_centroid_size);
    ReducedPartialAvgLabel<<<REDUCE_COUNT, arg->dims>>>(
        arg->input_len, arg->dims, arg->num_cluster, device_input, device_label,
        reduce_centroid, reduce_count);
    ReducedAvgLabel<<<arg->num_cluster, arg->dims>>>(
        arg->dims, arg->num_cluster, reduce_count, reduce_centroid,
        new_centroid);
    Converge<<<arg->num_cluster, 1>>>(
        arg->dims, arg->threshold, device_converge, old_centroid, new_centroid);
    hipMemcpy(converge, device_converge, converge_size,
               hipMemcpyDeviceToHost);
    bool converged = IsConverged(arg->num_cluster, converge);
    ++iteration;
    done = iteration >= arg->max_num_iter || converged;
  }
  TOCK(iteration);
  hipMemcpyAsync(label, device_label, label_size, hipMemcpyDeviceToHost);
  hipMemcpyAsync(centroid, new_centroid, centroid_size,
                  hipMemcpyDeviceToHost);
  hipDeviceSynchronize(); // must finish before free!
  hipFree(device_input);
  hipFree(device_centroid);
  hipFree(device_label);
  hipFree(device_converge);
  hipFree(reduce_centroid);
  hipFree(reduce_count);
  return KMeans_Ret{centroid, label};
}

int main(int argc, char *argv[]) {
  Arg arg = GetArg(argc, argv);
  KMeans_Ret ret = KMeans(&arg);
  if (arg.control_flag) {
    PrintCentroid(&arg, ret.centroid);
  } else {
    PrintLabel(&arg, ret.label);
  }
  free(ret.centroid);
  free(ret.label);
  FreeArg(&arg);
  return 0;
}